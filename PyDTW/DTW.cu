#include "hip/hip_runtime.h"
#include <ATen/Aten.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCAtomics.cuh>
#include <ATen/native/cuda/DTW.cuh>
#include <hip/hip_runtime.h>

namespace at{
    namespace native{
        //cuda kernel to compute the forward DTW cost matrix
        template <typename scalar_t>
        __global__ void compute_dtw_cuda_kernel(
            const scalar_t* __restrict__ D, //Input distance matrix
            scalar_t* __restrict__ R, //cost matrix to br computed
            int64_t max_i, //Number of rows in the distance matrix
            int64_t max_j, //Number of columns in the distance matrix
            int64_t B
        ){
            int64_t b = blockIdx.x; //batch index
            int64_t tid = threadIdx.x; //thread index
            int64_t I = tid; //row index

            for(int64_t j = 1; j<=max_j; j++){
                for(int64_t i = 1; i<=max_i; i++){
                    if(I==i){
                        //retrieve values from the previous cells in the cost matrix
                        scalar_t r0 = R[b *(max_i + 1)*(max_j + 1)+(i - 1)*(max_j + 1)+(j -1)];
                        scalar_t r1 = R[b *(max_i + 1)*(max_j + 1)+(i - 1)*(max_j + 1)+j];
                        scalar_t r2 = R[b*(max_i + 1)*(max_j + 1)+i*(max_j + 1)+(j-1)];

                        //compute the cost for the current cell
                        R[b*(max_i + 1)*(max_j + 1)+i*(max_j + 1)+j] = D[b*max_i*max_j+(i-1)*max_j+(j-1)]+min(r0, min(r1, r2));
                    }
                }
                __syncthreads(); //synchronize threads within a block
            }
        }

        //cuda kernel to compute the backward DTW Gradients
        template <typename scalar_t>
        __global__ void compute_dtw_backwards_cuda_kernel(
            const scalar_t* __restrict__ D, // Input distance matrix
            const scalar_t* __restrict__ R, // forward cost matrix
            scalar_t* __restrict__ E, //gradient matrix to be computed
            int64_t max_i, //number of rows in the distance matrix
            int64_t max_j, //number of columns in the distance matrix
            int64_t B //batch size
        ){
            int64_t b = blockIdx.x; // batch index
            int64_t tid = threadIdx.x; //thread index
            int64_t I = tid; //row index

            for(int64_t j = max_j; j>0; j--){
                for(int64_t i = max_i; i>0; i--){
                    if(I==i){
                        //compute the gradients for the current cell
                        scalar_t a =(R[b * (max_i + 1)*(max_j + 1)+(i+1)*(max_j + 1)+j] - R[b * (max_i + 1)*(max_j + 1)+i*(max_j + 1) + j] - D[b * max_i * max_j + max_j * i + j]);
                        scalar_t b =(R[b * (max_i + 1)*(max_j + 1)+i*(max_j + 1)+(j + 1)] - R[b*(max_i + 1)*(max_j + 1)+i*(max_j + 1) + j] - D[b * max_i * max_j + i * max_j + (j+1)]);
                        scalar_t c =(R[b * (max_i + 1)*(max_j + 1)+(i+1)*(max_j + 1)+(j+1)] - R[b*(max_i + 1)*(max_j + 1)+i*(max_j + 1)+j] - D[b*max_i*max_j+(i+1)*max_j+(j+1)]);

                        //update the gradient matrix
                        E[b*(max_i + 1) * (max_j + 1) + i * (max_j + 1) + j] = E[b * (max_i + 1) + (i + 1) * (max_j + 1) + j] * a + E[b * (max_i + 1) * (max_j + 1) + i * (max_j + 1) + (j + 1)] * b + E[b * (max_i + 1) + (max_j + 1) * (i+1) * (max_j + 1) + (j + 1)] * c;
                    }
                }
                __syncthreads(); //Synchronize threads within a block
            }
        }
        //Function to launch the forward DTW CUDA kernel
        Tensor dtw_forward_cuda(const Tensor& D){
            auto D_ = D.continous(); //Ensure contiguous memory layout
            auto dev = D.device().index(); //get device index
            auto dtype = D.scalar_type(); //get data type

            int64_t B = D.size(0); //batch size
            int64_t N = D.size(1); //number of rows
            int64_t M = D.size(2); //number of columns
            int threads_per_block = max(N, M); //number of threads per block

            //initialize the cost matrix with infinity
            auto R = at::empty({B, N+1, M+1}, D.options().dtype(dtype).device(D.device()));
            R.fill_(std::numeric_limits<float>::infinity());
            R.select(1, 0).fill_(0); // set the first row to zero
            R.select(2, 0).fill_(0); // set the first column to zero

            // launch the CUDA kernel for forward DTW
            AT_DISPATCH_FLOATING_TYPES(D.scalar_type(), "dtw_forward_cuda", ([&] {
                compute_dtw_cuda_kernel<scalar_t><<<B, threads_per_block>>>(
                    D_.data_ptr<scalar_t>(),
                    R.data_ptr<scalar_t>(),
                    N,
                    M,
                    B
                );
            }));

            //return the last element of the cost matrix (DTW distance)
            return R.select(1, N).select(1, M);
        }
        // Function to launch the backward DTW CUDA kernel
        Tensor dtw_backward_cuda(const Tensor& grad_output, const Tensor& D, const Tensor& R){
            auto D_ = D.contiguous(); //Ensure contiguous memory layout
            auto R_ = R.contiguous(); //Ensure contiguous memory layout
            auto dev = D.device().index(); //get device index
            auto dtype = D.scalar_type(); //get data type

            int64_t B = D.size(0); //batch size
            int64_t N = D.size(1); //number of rows
            int64_t M = D.size(2); //number of columns
            int threads_per_block = max(N, M); //number of threads per block

            //initialize the gradient matrix with zeros
            auto E = at::zeros({b, N+1, M+1}, D.options().dtype(dtype).device(D.device()));
            E.select(1, N).select(1, M).fill_(1); //set the last element to one

            // launch the CUDA kernel for backward DTW
            AT_DISPATCH_FLOATING_TYPES(D.scalar_type(), "dtw_backward_cuda", ([&] {
                compute_dtw_backwards_cuda_kernel<scalar_t><<<B, threads_per_block>>>(
                    D_.data_ptr<scalar_t>(),
                    R_.data_ptr<scalar_t>(),
                    E.data_ptr<scalar_t>(),
                    N,
                    M,
                    B
                );
            }));

            //return the gradient matrix excluding the padding
            return E.select(1,1).select(1,1);
        }
    } //namespace native
}   //namespace at